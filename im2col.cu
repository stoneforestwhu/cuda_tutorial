#include "hip/hip_runtime.h"
//  __global__ 表示共享内存，并行调用的关键字，gpu核函数必须以此声明
//  n 代表conv kernel的个数，n=Cin*H*W
//  DType*  data_im 代表一个张量，为输入图像(Cin,H,W)，以_im为后缀的都与输入图像相关
//  feature map的高宽和kernel的高宽
template <typename DType>
__global__ void im2col_gpu_kernel(const int n, 
                                  const DType* data_im,                                                                                                             const int height, const int width,             //  feature map的高宽
                                  const int kernel_h, const int kernel_w,        //  kernel的高宽
                                  const int pad_h, const int pad_w,                  
                                  const int stride_h, const int stride_w,            
                                  const int dilation_h, const int dilation_w,        
                                  const int height_col, const int width_col,     //  输出column的高宽
                                  DType* data_col)                               //  输出column张量，(Kh * Kw* Cin, H, W)
{     
    for(int index = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){ 
        //  在0-gridDim.x*blockDim.x之间并行，到n结束，index代表第i个conv kernel的索引
        //  index index of output matrix
        //  说明一下，若%某个维度的大小，表示要索引这个维度上的位置，而/这个维度的大小则要索引下个维度的位置
        //  这里说的下个维度，是从右向左。如这里的data_im，shape为(Cin,H,W)，那么顺序为W，H，Cin
        //  是总的索引，%是在某个维度上某个范围内的索引
        const int h_index = index / width_col;
        const int h_col = h_index % height_col; // // 在某个c_in的维度下高的索引
        const int w_col = index % width_col;    // 在某个高的维度下宽的索引
        const int c_im = h_index / height_col;  // 输入通道索引
        const int c_col = c_im * kernel_h * kernel_w; // 输出通道索引
        const int h_offset = h_col * stride_h - pad_h; // 输出h的偏移
        const int w_offset = w_col * stride_w - pad_w; // 输出w的偏移
        DType* data_col_ptr = data_col;  //获得输出张量的指针拷贝
        // 指针向前移动，由于index是0-Cin*H*W，c_col,h_col和w_col有Cin、H和W种取值，正好对应index
        data_col_ptr += (c_col * height_col + h_col) * width_col + w_col; 
        const DType* data_im_ptr = data_im; //  获取输入张量的指针拷贝
        data_im_ptr += (c_im * height + h_offset) * width + w_offset; //指针向前移动
        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) { // 对单个kernel进行循环
                int h_im = h_offset + i * dilation_h;
                int w_im = w_offset + j * dilation_w;
                // *+指针是只取指针所指位置的数值，这里赋值给对应位置
                *data_col_ptr = (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ? // 若索引不越界
                                data_im_ptr[i * dilation_h * width + j * dilation_w] : 
                                static_cast<DType>(0);
                data_col_ptr += height_col * width_col;
            }
        }
}
